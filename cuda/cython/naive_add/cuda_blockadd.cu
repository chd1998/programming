#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"

__global__ void blockadd(float *a, float *b, float *out, int n) {
    int tid = blockIdx.x;
    if (tid < n)
        out[tid] = a[tid] + b[tid];
}

extern "C" int blockmain(float *a, float *b, float *out, int n) {
    float *dev_a, *dev_b, *dev_out;

    // allocate the memory on the GPU
    hipMalloc((void**)&dev_a, n*sizeof(float));
    hipMalloc((void**)&dev_b, n*sizeof(float));
    hipMalloc((void**)&dev_out, n*sizeof(float));

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    blockadd<<<n,1>>>(dev_a, dev_b, dev_out, n);

    // copy the array 'out' back from the GPU to the CPU
    hipMemcpy(out, dev_out, n * sizeof(float), hipMemcpyDeviceToHost);

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_out);

    return 0;
}
