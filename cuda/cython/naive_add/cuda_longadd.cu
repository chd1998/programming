#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"

__global__ void longadd(float *a, float *b, float *out, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n){
        out[tid] = a[tid] + b[tid];
        tid += gridDim.x * blockDim.x;
    }
}

extern "C" int longmain(float *a, float *b, float *out, int n) {
    float *dev_a, *dev_b, *dev_out;

    // allocate the memory on the GPU
    hipMalloc((void**)&dev_a, n*sizeof(float));
    hipMalloc((void**)&dev_b, n*sizeof(float));
    hipMalloc((void**)&dev_out, n*sizeof(float));

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    longadd<<<256, 256>>>(dev_a, dev_b, dev_out, n);

    // copy the array 'out' back from the GPU to the CPU
    hipMemcpy(out, dev_out, n * sizeof(float), hipMemcpyDeviceToHost);

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_out);

    return 0;
}
